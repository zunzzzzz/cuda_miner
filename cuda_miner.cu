#include "hip/hip_runtime.h"
//***********************************************************************************
// 2018.04.01 created by Zexlus1126
//
//    Example 002
// This is a simple demonstration on calculating merkle root from merkle branch 
// and solving a block (#286819) which the information is downloaded from Block Explorer 
//***********************************************************************************

#include <iostream>
#include <fstream>
#include <string>

#include <cstdio>
#include <cstring>
#include <ctime>

#include <cassert>

#include "sha256.h"

////////////////////////   Block   /////////////////////

typedef struct _block
{
    unsigned int version;
    unsigned char prevhash[32];
    unsigned char merkle_root[32];
    unsigned int ntime;
    unsigned int nbits;
    unsigned int nonce;
}HashBlock;

/////////////////// Global variable in GPU////////////////
__device__ HashBlock block_gpu;
__device__ SHA256 sha256_ctx_gpu;
__device__ bool found = false;

////////////////////////   Utils   ///////////////////////

//convert one hex-codec char to binary
unsigned char decode(unsigned char c)
{
    switch(c)
    {
        case 'a':
            return 0x0a;
        case 'b':
            return 0x0b;
        case 'c':
            return 0x0c;
        case 'd':
            return 0x0d;
        case 'e':
            return 0x0e;
        case 'f':
            return 0x0f;
        case '0' ... '9':
            return c-'0';
        default:
            return 0x0a;
    }
}


// convert hex string to binary
//
// in: input string
// string_len: the length of the input string
//      '\0' is not included in string_len!!!
// out: output bytes array
void convert_string_to_little_endian_bytes(unsigned char* out, char *in, size_t string_len)
{
    assert(string_len % 2 == 0);

    size_t s = 0;
    size_t b = string_len/2-1;

    for(; s < string_len; s+=2, --b)
    {
        out[b] = (unsigned char)(decode(in[s])<<4) + decode(in[s+1]);
    }
}

// print out binary array (from highest value) in the hex format
void print_hex(unsigned char* hex, size_t len)
{
    for(int i=0;i<len;++i)
    {
        printf("%02x", hex[i]);
    }
}


// print out binar array (from lowest value) in the hex format
void print_hex_inverse(unsigned char* hex, size_t len)
{
    for(int i=len-1;i>=0;--i)
    {
        printf("%02x", hex[i]);
    }
}
__device__
void print_hex_inverse_gpu(unsigned char* hex, size_t len)
{
    for(int i=len-1;i>=0;--i)
    {
        printf("%02x", hex[i]);
    }
}
int little_endian_bit_comparison(const unsigned char *a, const unsigned char *b, size_t byte_len)
{
    // compared from lowest bit
    for(int i=byte_len-1;i>=0;--i)
    {
        if(a[i] < b[i])
            return -1;
        else if(a[i] > b[i])
            return 1;
    }
    return 0;
}
__device__
int little_endian_bit_comparison_gpu(const unsigned char *a, const unsigned char *b, size_t byte_len)
{
    // compared from lowest bit
    for(int i=byte_len-1;i>=0;--i)
    {
        if(a[i] < b[i])
            return -1;
        else if(a[i] > b[i])
            return 1;
    }
    return 0;
}
void getline(char *str, size_t len, FILE *fp)
{

    int i=0;
    while( i<len && (str[i] = fgetc(fp)) != EOF && str[i++] != '\n');
    str[len-1] = '\0';
}

////////////////////////   Hash   ///////////////////////

void double_sha256_cpu(SHA256 *sha256_ctx, unsigned char *bytes, size_t len)
{
    SHA256 tmp;
    sha256_cpu(&tmp, (BYTE*)bytes, len);
    sha256_cpu(sha256_ctx, (BYTE*)&tmp, sizeof(tmp));
}
__device__
void double_sha256_gpu(SHA256 *sha256_ctx, unsigned char *bytes, size_t len)
{
    SHA256 tmp;
    sha256_gpu(&tmp, (BYTE*)bytes, len);
    sha256_gpu(sha256_ctx, (BYTE*)&tmp, sizeof(tmp));
}

////////////////////   Merkle Root   /////////////////////


// calculate merkle root from several merkle branches
// root: output hash will store here (little-endian)
// branch: merkle branch  (big-endian)
// count: total number of merkle branch
void calc_merkle_root(unsigned char *root, int count, char **branch)
{
    size_t total_count = count; // merkle branch
    unsigned char *raw_list = new unsigned char[(total_count+1)*32];
    unsigned char **list = new unsigned char*[total_count+1];

    // copy each branch to the list
    for(int i=0;i<total_count; ++i)
    {
        list[i] = raw_list + i * 32;
        //convert hex string to bytes array and store them into the list
        convert_string_to_little_endian_bytes(list[i], branch[i], 64);
    }

    list[total_count] = raw_list + total_count*32;


    // calculate merkle root
    while(total_count > 1)
    {
        
        // hash each pair
        int i, j;

        if(total_count % 2 == 1)  //odd, 
        {
            memcpy(list[total_count], list[total_count-1], 32);
        }

        for(i=0, j=0;i<total_count;i+=2, ++j)
        {
            // this part is slightly tricky,
            //   because of the implementation of the double_sha256,
            //   we can avoid the memory begin overwritten during our sha256d calculation
            // double_sha:
            //     tmp = hash(list[0]+list[1])
            //     list[0] = hash(tmp)
            double_sha256_cpu((SHA256*)list[j], list[i], 64);
        }

        total_count = j;
    }

    memcpy(root, list[0], 32);

    delete[] raw_list;
    delete[] list;
}
__global__
void FindNonce(unsigned char* target_hex)
{   
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    // printf("%d %d %d %d\n", blockDim.x, gridDim.x, stride, blockIdx.x);
    HashBlock block_tmp = block_gpu;
    SHA256 sha256_ctx_tmp = sha256_ctx_gpu;
    __syncthreads();
    for(block_tmp.nonce = index; block_tmp.nonce<=0xffffffff && found == false; block_tmp.nonce += stride)
    {   
        //sha256d
        double_sha256_gpu(&sha256_ctx_tmp, (unsigned char*)&block_tmp, sizeof(block_tmp));
        if(block_tmp.nonce % 1000000 == 0)
        {
            // printf("index : %d,", index);
            // printf("hash #%10u (big): ", block_tmp.nonce);
            // print_hex_inverse_gpu(sha256_ctx_tmp.b, 32);
            // printf("\n");
        }
        
        if(little_endian_bit_comparison_gpu(sha256_ctx_tmp.b, target_hex, 32) < 0)  // sha256_ctx < target_hex
        {
            printf("Found Solution!!\n");
            printf("hash #%10u (big): ", block_tmp.nonce);
            print_hex_inverse_gpu(sha256_ctx_tmp.b, 32);
            printf("\n\n");
            found = true;
            block_gpu = block_tmp;
            sha256_ctx_gpu = sha256_ctx_tmp;
        }
    }
}
void solve(FILE *fin, FILE *fout)
{
    clock_t start, stop;  
    
    // **** read data *****
    char version[9];
    char prevhash[65];
    char ntime[9];
    char nbits[9];
    int tx;
    char *raw_merkle_branch;
    char **merkle_branch;

    getline(version, 9, fin);
    getline(prevhash, 65, fin);
    getline(ntime, 9, fin);
    getline(nbits, 9, fin);
    fscanf(fin, "%d\n", &tx);

    raw_merkle_branch = new char [tx * 65];
    merkle_branch = new char *[tx];
    for(int i=0;i<tx;++i)
    {
        merkle_branch[i] = raw_merkle_branch + i * 65;
        getline(merkle_branch[i], 65, fin);
        merkle_branch[i][64] = '\0';
    }
    // **** calculate merkle root ****
    unsigned char merkle_root[32];
    calc_merkle_root(merkle_root, tx, merkle_branch);

    printf("merkle root(little): ");
    print_hex(merkle_root, 32);
    printf("\n");

    printf("merkle root(big):    ");
    print_hex_inverse(merkle_root, 32);
    printf("\n");


    // **** solve block ****
    printf("Block info (big): \n");
    printf("  version:  %s\n", version);
    printf("  pervhash: %s\n", prevhash);
    printf("  merkleroot: "); print_hex_inverse(merkle_root, 32); printf("\n");
    printf("  nbits:    %s\n", nbits);
    printf("  ntime:    %s\n", ntime);
    printf("  nonce:    ???\n\n");

    HashBlock block;
    


    // convert to byte array in little-endian
    convert_string_to_little_endian_bytes((unsigned char *)&block.version, version, 8);
    convert_string_to_little_endian_bytes(block.prevhash,                  prevhash,    64);
    memcpy(block.merkle_root, merkle_root, 32);
    convert_string_to_little_endian_bytes((unsigned char *)&block.nbits,   nbits,     8);
    convert_string_to_little_endian_bytes((unsigned char *)&block.ntime,   ntime,     8);
    block.nonce = 0;
    
    
    // ********** calculate target value *********
    // calculate target value from encoded difficulty which is encoded on "nbits"
    unsigned int exp = block.nbits >> 24;
    unsigned int mant = block.nbits & 0xffffff;
    // unsigned char target_hex[32] = {};
    unsigned char* target_hex;
    size_t size= sizeof(unsigned char) * 32;
    hipMallocManaged(&target_hex, size);

    unsigned int shift = 8 * (exp - 3);
    unsigned int sb = shift / 8;
    unsigned int rb = shift % 8;
    
    // little-endian
    target_hex[sb    ] = (mant << rb);
    target_hex[sb + 1] = (mant >> (8-rb));
    target_hex[sb + 2] = (mant >> (16-rb));
    target_hex[sb + 3] = (mant >> (24-rb));
    
    
    printf("Target value (big): ");
    print_hex_inverse(target_hex, 32);
    printf("\n");

    // ********** find nonce **************
    start = clock();
    
    size_t threads_per_block = 128;
    size_t num_of_blocks = 50 * 3;
    bool is_found = false;
    SHA256 sha256_ctx;
    hipMemcpyToSymbol(HIP_SYMBOL(block_gpu), &block, sizeof(HashBlock), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(sha256_ctx_gpu), &sha256_ctx, sizeof(SHA256), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(found), &is_found, sizeof(bool), 0, hipMemcpyHostToDevice);
    FindNonce<<<num_of_blocks, threads_per_block>>>(&target_hex[0]);
    hipMemcpyFromSymbol(&block, HIP_SYMBOL(block_gpu), sizeof(HashBlock), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&sha256_ctx, HIP_SYMBOL(sha256_ctx_gpu), sizeof(SHA256), 0, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    stop = clock();
    std::cout << (stop - start) / CLOCKS_PER_SEC << std::endl;
    // print result

    //little-endian
    printf("hash(little): ");
    print_hex(sha256_ctx.b, 32);
    printf("\n");

    //big-endian
    printf("hash(big):    ");
    print_hex_inverse(sha256_ctx.b, 32);
    printf("\n\n");

    for(int i=0;i<4;++i)
    {
        fprintf(fout, "%02x", ((unsigned char*)&block.nonce)[i]);
    }
    fprintf(fout, "\n");

    delete[] merkle_branch;
    delete[] raw_merkle_branch;
    hipDeviceReset();
}

int main(int argc, char **argv)
{
    if (argc != 3) {
        fprintf(stderr, "usage: cuda_miner <in> <out>\n");
    }
    FILE *fin = fopen(argv[1], "r");
    FILE *fout = fopen(argv[2], "w");

    int totalblock;

    fscanf(fin, "%d\n", &totalblock);
    fprintf(fout, "%d\n", totalblock);

    for(int i=0;i<totalblock;++i)
    {
        solve(fin, fout);
    }

    return 0;
}

